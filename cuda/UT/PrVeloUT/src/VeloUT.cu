#include "hip/hip_runtime.h"
#include "VeloUT.cuh"

__global__ void veloUT(
  uint* dev_ut_hits,
  uint* dev_ut_hit_count,
  int* dev_atomics_storage,
  uint* dev_velo_track_hit_number,
  uint* dev_velo_track_hits,
  uint* dev_velo_states,
  VeloUTTracking::TrackUT* dev_veloUT_tracks,
  int* dev_atomics_veloUT,
  PrUTMagnetTool* dev_ut_magnet_tool,
  float* dev_ut_dxDy
) {
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const uint total_number_of_hits = dev_ut_hit_count[number_of_events * VeloUTTracking::n_layers];
  
  // Velo consolidated types
  const Velo::Consolidated::Tracks velo_tracks {(uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  const Velo::Consolidated::States velo_states {dev_velo_states, velo_tracks.total_number_of_tracks};
  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  UTHitCount ut_hit_count;
  ut_hit_count.typecast_after_prefix_sum(dev_ut_hit_count, event_number, number_of_events);

  UTHits ut_hits;
  ut_hits.typecast_sorted(dev_ut_hits, total_number_of_hits);

  /* dev_atomics_veloUT contains in an SoA:
     1. # of veloUT tracks
     2. # velo tracks in UT acceptance
  */
  int* n_veloUT_tracks_event = dev_atomics_veloUT + event_number;
  VeloUTTracking::TrackUT* veloUT_tracks_event = dev_veloUT_tracks + event_number * VeloUTTracking::max_num_tracks;
  int* n_velo_tracks_in_UT_event = dev_atomics_veloUT + number_of_events + event_number;
  
  // initialize atomic veloUT tracks counter
  if ( threadIdx.x == 0 ) {
    *n_veloUT_tracks_event = 0;
    *n_velo_tracks_in_UT_event = 0;
  }
  __syncthreads();

  int posLayers[4][85];

  // printf("first hit: cos = %f, yBegin = %f, yEnd = %f, zAtYEq0 = %f, xAtYEq0 = %f, weight = %f, highThreshold = %u \n",
  //        hits_layers_event->cos(0),
  //        hits_layers_event->yBegin(0),
  //        hits_layers_event->yEnd(0),
  //        hits_layers_event->zAtYEq0(0),
  //        hits_layers_event->xAtYEq0(0),
  //        hits_layers_event->weight(0),
  //        hits_layers_event->highThreshold(0));
         
  fillIterators(ut_hits, ut_hit_count, posLayers);

  const float* fudgeFactors = &(dev_ut_magnet_tool->dxLayTable[0]);
  const float* bdlTable     = &(dev_ut_magnet_tool->bdlTable[0]);

  // array to store indices of selected hits in layers
  // -> can then access the hit information in the HitsSoA
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers];
  
  for ( int i = 0; i < (number_of_tracks_event + blockDim.x - 1) / blockDim.x; ++i) {
    const int i_track = i * blockDim.x + threadIdx.x;
    
    const uint velo_states_index = event_tracks_offset + i_track;
    if (i_track >= number_of_tracks_event) continue;
    if (velo_states.backward[velo_states_index]) continue;

    // Mini State with only x, y, tx, ty and z
    MiniState velo_state {velo_states, velo_states_index};

    if(!veloTrackInUTAcceptance(velo_state)) continue;

    atomicAdd(n_velo_tracks_in_UT_event, 1);

     // for storing calculated x position of hits for this track
    float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer];

    for ( int i_layer = 0; i_layer < VeloUTTracking::n_layers; ++i_layer ) {
      n_hitCandidatesInLayers[i_layer] = 0;
    }

    if( !getHits(
          hitCandidatesInLayers,
          n_hitCandidatesInLayers,
          x_pos_layers,
          posLayers,
          ut_hits,
          ut_hit_count,
          fudgeFactors,
          velo_state,
          dev_ut_dxDy)
        ) continue;

    TrackHelper helper {velo_state};

    // indices within hitCandidatesInLayers for selected hits belonging to best track 
    int hitCandidateIndices[VeloUTTracking::n_layers];
    
    // go through UT layers in forward direction
    if(!formClusters(
          hitCandidatesInLayers,
          n_hitCandidatesInLayers,
          x_pos_layers,
          hitCandidateIndices,
          ut_hits,
          ut_hit_count,
          helper,
          velo_state,
          dev_ut_dxDy,
          true)) {
      
      // go through UT layers in backward direction
      formClusters(
        hitCandidatesInLayers,
        n_hitCandidatesInLayers,
        x_pos_layers,
        hitCandidateIndices,
        ut_hits,
        ut_hit_count,
        helper,
        velo_state,
        dev_ut_dxDy,
        false);
    }
    
    if ( helper.n_hits > 0 ) {
      const uint velo_track_hit_number = velo_tracks.number_of_hits(i_track);
      const Velo::Consolidated::Hits velo_track_hits = velo_tracks.get_hits(dev_velo_track_hits, i_track);

      prepareOutputTrack(
        velo_track_hits,
        velo_track_hit_number,
        helper,
        velo_state,
        hitCandidatesInLayers,
        n_hitCandidatesInLayers,
        ut_hits,
        ut_hit_count,
        x_pos_layers,
        hitCandidateIndices,
        veloUT_tracks_event,
        n_veloUT_tracks_event,
        bdlTable);
    }
    
  } // velo tracks
 
}
