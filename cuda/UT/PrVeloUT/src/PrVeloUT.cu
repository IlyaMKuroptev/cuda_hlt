#include "hip/hip_runtime.h"
#include "PrVeloUT.cuh"

//-----------------------------------------------------------------------------
// Implementation file for PrVeloUT
//
// 2007-05-08: Mariusz Witek
// 2017-03-01: Christoph Hasse (adapt to future framework)
// 2018-05-05: Plácido Fernández (make standalone)
// 2018-07:    Dorothea vom Bruch (convert to C and then CUDA code)
//-----------------------------------------------------------------------------

// -- These things are all hardcopied from the PrTableForFunction
// -- and PrUTMagnetTool
// -- If the granularity or whatever changes, this will give wrong results
__host__ __device__ int masterIndex(const int index1, const int index2, const int index3){
  return (index3*11 + index2)*31 + index1;
}

//=============================================================================
// Reject tracks outside of acceptance or pointing to the beam pipe
//=============================================================================
__host__ __device__ bool veloTrackInUTAcceptance(
  const MiniState& state
) {
  const float xMidUT = state.x + state.tx*( PrVeloUTConst::zMidUT - state.z);
  const float yMidUT = state.y + state.ty*( PrVeloUTConst::zMidUT - state.z);

  if( xMidUT*xMidUT+yMidUT*yMidUT  < PrVeloUTConst::centralHoleSize*PrVeloUTConst::centralHoleSize ) return false;
  if( (std::abs(state.tx) > PrVeloUTConst::maxXSlope) || (std::abs(state.ty) > PrVeloUTConst::maxYSlope) ) return false;

  if(PrVeloUTConst::passTracks && std::abs(xMidUT) < PrVeloUTConst::passHoleSize && std::abs(yMidUT) < PrVeloUTConst::passHoleSize) {
    return false;
  }

  return true;
}

//=============================================================================
// Find the hits
//=============================================================================
__host__ __device__ bool getHits(
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  const int posLayers[4][85],
  UTHits& ut_hits,
  UTHitCount& ut_hit_count,
  const float* fudgeFactors, 
  const MiniState& trState,
  const float* ut_dxDy)
{
  // -- This is hardcoded, so faster
  // -- If you ever change the Table in the magnet tool, this will be wrong
  const float absSlopeY = std::abs( trState.ty );
  const int index = (int)(absSlopeY*100 + 0.5);
  assert( 3 + 4*index < PrUTMagnetTool::N_dxLay_vals );
  const std::array<float,4> normFact = { 
    fudgeFactors[4*index], 
    fudgeFactors[1 + 4*index], 
    fudgeFactors[2 + 4*index], 
    fudgeFactors[3 + 4*index] 
  };

  // -- this 500 seems a little odd...
  // to do: change back!
  const float invTheta = std::min(500., 1.0/std::sqrt(trState.tx*trState.tx+trState.ty*trState.ty));
  //const float minMom   = std::max(PrVeloUTConst::minPT*invTheta, PrVeloUTConst::minMomentum);
  const float minMom   = std::max(PrVeloUTConst::minPT*invTheta, float(1.5)*Gaudi::Units::GeV);
  const float xTol     = std::abs(1. / ( PrVeloUTConst::distToMomentum * minMom ));
  const float yTol     = PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * xTol;

  int nLayers = 0;

  float dxDyHelper[VeloUTTracking::n_layers] = {0., 1., -1., 0};
  for(int iStation = 0; iStation < 2; ++iStation) {

    if( iStation == 1 && nLayers == 0 ) return false;

    for(int iLayer = 0; iLayer < 2; ++iLayer) {
      if( iStation == 1 && iLayer == 1 && nLayers < 2 ) return false;

      int layer = 2*iStation+iLayer;
      int layer_offset = ut_hit_count.layer_offsets[layer];
      
      if( ut_hit_count.n_hits_layers[layer] == 0 ) continue;
      const float dxDy   = ut_dxDy[layer];
      const float zLayer = ut_hits.zAtYEq0[layer_offset + 0]; 

      const float yAtZ   = trState.y + trState.ty*(zLayer - trState.z);
      const float xLayer = trState.x + trState.tx*(zLayer - trState.z);
      const float yLayer = yAtZ + yTol * dxDyHelper[layer];

      const float normFactNum = normFact[2*iStation + iLayer];
      const float invNormFact = 1.0/normFactNum;

      const float lowerBoundX =
        (xLayer - dxDy*yLayer) - xTol*invNormFact - std::abs(trState.tx)*PrVeloUTConst::intraLayerDist;
      const float upperBoundX =
        (xLayer - dxDy*yLayer) + xTol*invNormFact + std::abs(trState.tx)*PrVeloUTConst::intraLayerDist;

      const int indexLowProto = lowerBoundX > 0 ? std::sqrt( std::abs(lowerBoundX)*2.0 ) + 42 : 42 - std::sqrt( std::abs(lowerBoundX)*2.0 );
      const int indexHiProto  = upperBoundX > 0 ? std::sqrt( std::abs(upperBoundX)*2.0 ) + 43 : 43 - std::sqrt( std::abs(upperBoundX)*2.0 );

      const int indexLow  = std::max( indexLowProto, 0 );
      const int indexHi   = std::min( indexHiProto, 84);

      size_t posBeg = posLayers[layer][ indexLow ];
      size_t posEnd = posLayers[layer][ indexHi  ];

      while ( (ut_hits.xAtYEq0[layer_offset + posBeg] < lowerBoundX) && (posBeg != ut_hit_count.n_hits_layers[layer] ) ) {
        ++posBeg;
      }
      
      if (posBeg == ut_hit_count.n_hits_layers[layer]) continue;

      findHits(posBeg, posEnd,
        ut_hits, layer_offset, layer, ut_dxDy,
        trState, xTol*invNormFact, invNormFact,
        hitCandidatesInLayers[layer], n_hitCandidatesInLayers[layer],
        x_pos_layers);

      nLayers += int( !( n_hitCandidatesInLayers[layer] == 0 ) );
    }
  }

  return nLayers > 2;
}

//=========================================================================
// Form clusters
//=========================================================================
__host__ __device__ bool formClusters(
  const int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  const int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  const float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int bestHitCandidateIndices[VeloUTTracking::n_layers],
  UTHits& ut_hits,
  UTHitCount& ut_hit_count,
  TrackHelper& helper,
  MiniState& state,
  const float* ut_dxDy,
  const bool forward)
{
  // handle forward / backward cluster search
  int layers[VeloUTTracking::n_layers];
  for ( int i_layer = 0; i_layer < VeloUTTracking::n_layers; ++i_layer ) {
      if ( forward )
        layers[i_layer] = i_layer;
      else
        layers[i_layer] = VeloUTTracking::n_layers - 1 - i_layer;
  }


  // Go through the layers
  bool fourLayerSolution = false;
  int hitCandidateIndices[VeloUTTracking::n_layers];
  for ( int i_hit0 = 0; i_hit0 < n_hitCandidatesInLayers[ layers[0] ]; ++i_hit0 ) {

    const int layer_offset0 = ut_hit_count.layer_offsets[ layers[0] ];
    const int hit_index0    = layer_offset0 + hitCandidatesInLayers[ layers[0] ][i_hit0];
    const float xhitLayer0  = x_pos_layers[layers[0]][i_hit0];
    const float zhitLayer0  = ut_hits.zAtYEq0[hit_index0];
    hitCandidateIndices[0] = i_hit0;
    
    for ( int i_hit2 = 0; i_hit2 < n_hitCandidatesInLayers[ layers[2] ]; ++i_hit2 ) {

      const int layer_offset2 = ut_hit_count.layer_offsets[ layers[2] ];
      const int hit_index2    = layer_offset2 + hitCandidatesInLayers[ layers[2] ][i_hit2];
      const float xhitLayer2  = x_pos_layers[layers[2]][i_hit2];
      const float zhitLayer2  = ut_hits.zAtYEq0[hit_index2];
      hitCandidateIndices[2] = i_hit2;
      
      const float tx = (xhitLayer2 - xhitLayer0)/(zhitLayer2 - zhitLayer0);
      if( std::abs(tx-state.tx) > PrVeloUTConst::deltaTx2 ) continue;
            
      int IndexBestHit1 = -10;
      float hitTol = PrVeloUTConst::hitTol2;
      for ( int i_hit1 = 0; i_hit1 < n_hitCandidatesInLayers[ layers[1] ]; ++i_hit1 ) {

        const int layer_offset1 = ut_hit_count.layer_offsets[ layers[1] ];
        const int hit_index1    = layer_offset1 + hitCandidatesInLayers[ layers[1] ][i_hit1];
        const float xhitLayer1  = x_pos_layers[layers[1]][i_hit1];
        const float zhitLayer1  = ut_hits.zAtYEq0[hit_index1];
       
        const float xextrapLayer1 = xhitLayer0 + tx*(zhitLayer1-zhitLayer0);
        if(std::abs(xhitLayer1 - xextrapLayer1) < hitTol){
          hitTol = std::abs(xhitLayer1 - xextrapLayer1);
          IndexBestHit1 = hit_index1;
          hitCandidateIndices[1] = i_hit1;
        }
      } // loop over layer 1
      
      if( fourLayerSolution && IndexBestHit1 < 0 ) continue;

      int IndexBestHit3 = -10;
      hitTol = PrVeloUTConst::hitTol2;
      for ( int i_hit3 = 0; i_hit3 < n_hitCandidatesInLayers[ layers[3] ]; ++i_hit3 ) {

        const int layer_offset3 = ut_hit_count.layer_offsets[ layers[3] ];
        const int hit_index3    = layer_offset3 + hitCandidatesInLayers[ layers[3] ][i_hit3];
        const float xhitLayer3  = x_pos_layers[layers[3]][i_hit3];
        const float zhitLayer3  = ut_hits.zAtYEq0[hit_index3];
        
        const float xextrapLayer3 = xhitLayer2 + tx*(zhitLayer3-zhitLayer2);
        if(std::abs(xhitLayer3 - xextrapLayer3) < hitTol){
          hitTol = std::abs(xhitLayer3 - xextrapLayer3);
          IndexBestHit3 = hit_index3;
          hitCandidateIndices[3] = i_hit3;
        }
      } // loop over layer 3
     
      // -- All hits found
      if ( IndexBestHit1 > 0 && IndexBestHit3 > 0 ) {
        const int hitIndices[4] = {hit_index0, IndexBestHit1, hit_index2, IndexBestHit3};
        simpleFit<4>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        
        if(!fourLayerSolution && helper.n_hits > 0){
          fourLayerSolution = true;
        }
        continue;
      }

      // -- Nothing found in layer 3
      if( !fourLayerSolution && IndexBestHit1 > 0 ){
        const int hitIndices[3] = {hit_index0, IndexBestHit1, hit_index2};
        simpleFit<3>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        continue;
      }
      // -- Nothing found in layer 1
      if( !fourLayerSolution && IndexBestHit3 > 0 ){
        hitCandidateIndices[1] = hitCandidateIndices[3];  // hit3 saved in second position of hits4fit
        const int hitIndices[3] = {hit_index0, IndexBestHit3, hit_index2};
        simpleFit<3>(x_pos_layers, hitCandidateIndices, bestHitCandidateIndices, hitCandidatesInLayers, ut_hits, hitIndices, helper, state, ut_dxDy);
        continue;
      }
      
    }
  }

  return fourLayerSolution;
}
//=========================================================================
// Create the Velo-UT tracks
//=========================================================================
__host__ __device__ void prepareOutputTrack(
  const Velo::Consolidated::Hits& velo_track_hits,
  const uint velo_track_hit_number,
  const TrackHelper& helper,
  const MiniState& state,
  int hitCandidatesInLayers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  int n_hitCandidatesInLayers[VeloUTTracking::n_layers],
  UTHits& ut_hits,
  UTHitCount& ut_hit_count,
  const float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer],
  const int hitCandidateIndices[VeloUTTracking::n_layers],
  VeloUTTracking::TrackUT VeloUT_tracks[VeloUTTracking::max_num_tracks],
  int* n_veloUT_tracks,
  const float* bdlTable) {

  //== Handle states. copy Velo one, add UT.
  const float zOrigin = (std::fabs(state.ty) > 0.001)
    ? state.z - state.y / state.ty
    : state.z - state.x / state.tx;

  // -- These are calculations, copied and simplified from PrTableForFunction
  const std::array<float,3> var = { state.ty, zOrigin, state.z };

  const int index1 = std::max(0, std::min( 30, int((var[0] + 0.3)/0.6*30) ));
  const int index2 = std::max(0, std::min( 10, int((var[1] + 250)/500*10) ));
  const int index3 = std::max(0, std::min( 10, int( var[2]/800*10)        ));

  assert( masterIndex(index1, index2, index3) < PrUTMagnetTool::N_bdl_vals );
  float bdl = bdlTable[masterIndex(index1, index2, index3)];

  const float bdls[3] = { bdlTable[masterIndex(index1+1, index2,index3)],
                          bdlTable[masterIndex(index1,index2+1,index3)],
                          bdlTable[masterIndex(index1,index2,index3+1)] };
  const float deltaBdl[3]   = { 0.02, 50.0, 80.0 };
  const float boundaries[3] = { -0.3f + float(index1)*deltaBdl[0],
                                -250.0f + float(index2)*deltaBdl[1],
                                0.0f + float(index3)*deltaBdl[2] };

  // -- This is an interpolation, to get a bit more precision
  float addBdlVal = 0.0;
  const float minValsBdl[3] = { -0.3, -250.0, 0.0 };
  const float maxValsBdl[3] = { 0.3, 250.0, 800.0 };
  for(int i=0; i<3; ++i) {
    if( var[i] < minValsBdl[i] || var[i] > maxValsBdl[i] ) continue;
    const float dTab_dVar =  (bdls[i] - bdl) / deltaBdl[i];
    const float dVar = (var[i]-boundaries[i]);
    addBdlVal += dTab_dVar*dVar;
  }
  bdl += addBdlVal;
  // ----

  const float qpxz2p =-1*std::sqrt(1.+state.ty*state.ty)/bdl*3.3356/Gaudi::Units::GeV;
  const float qop = (std::abs(bdl) < 1.e-8) ? 0.0 : helper.bestParams[0]*qpxz2p;

  // -- Don't make tracks that have grossly too low momentum
  // -- Beware of the momentum resolution!
  const float p  = 1.3*std::abs(1/qop);
  const float pt = p*std::sqrt(state.tx*state.tx + state.ty*state.ty);

  if( p < PrVeloUTConst::minMomentum || pt < PrVeloUTConst::minPT ) return;

#ifdef __CUDA_ARCH__
  uint n_tracks = atomicAdd(n_veloUT_tracks, 1);
#else
  (*n_veloUT_tracks)++;
  uint n_tracks = *n_veloUT_tracks - 1;
#endif

  
  const float txUT = helper.bestParams[3];

  // TODO: Maybe have a look and optimize this if possible
  VeloUTTracking::TrackUT track;
  track.hitsNum = 0;
  for (int i=0; i<velo_track_hit_number; ++i) {
    track.addLHCbID(velo_track_hits.LHCbID[i]);
    assert( track.hitsNum < VeloUTTracking::max_track_size);
  }
  track.set_qop( qop );
  
  // Adding overlap hits
  for ( int i_hit = 0; i_hit < helper.n_hits; ++i_hit ) {
    const int hit_index = helper.bestHitIndices[i_hit];
    
    track.addLHCbID( ut_hits.LHCbID[hit_index] );
    assert( track.hitsNum < VeloUTTracking::max_track_size);
    
    const int planeCode = ut_hits.planeCode[hit_index];
    const float xhit = x_pos_layers[ planeCode ][ hitCandidateIndices[i_hit] ];
    const float zhit = ut_hits.zAtYEq0[hit_index];

    const int layer_offset = ut_hit_count.layer_offsets[ planeCode ];
    for ( int i_ohit = 0; i_ohit < n_hitCandidatesInLayers[planeCode]; ++i_ohit ) {
      const int ohit_index = hitCandidatesInLayers[planeCode][i_ohit];
      const float zohit  = ut_hits.zAtYEq0[layer_offset + ohit_index];
      
      if(zohit==zhit) continue;
      
      const float xohit = x_pos_layers[ planeCode ][ i_ohit];
      const float xextrap = xhit + txUT*(zhit-zohit);
      if( xohit-xextrap < -PrVeloUTConst::overlapTol) continue;
      if( xohit-xextrap > PrVeloUTConst::overlapTol) break;
      
      track.addLHCbID(ut_hits.LHCbID[layer_offset + ohit_index]);
      assert( track.hitsNum < VeloUTTracking::max_track_size);
      
      // -- only one overlap hit
      break;
    }
  }
  assert( n_tracks < VeloUTTracking::max_num_tracks );
  VeloUT_tracks[n_tracks] = track;

  /*
  outTr.x = state.x;
  outTr.y = state.y;
  outTr.z = state.z;
  outTr.tx = state.tx;
  outTr.ty = state.ty;
  */
}

__host__ __device__ void fillArray(
  int * array,
  const int size,
  const size_t value ) {
  for ( int i = 0; i < size; ++i ) {
    array[i] = value;
  }
}

__host__ __device__ void fillArrayAt(
  int * array,
  const int offset,
  const int n_vals,
  const size_t value ) {  
    fillArray( array + offset, n_vals, value ); 
}

// ==============================================================================
// -- Method to cache some starting points for the search
// -- This is actually faster than binary searching the full array
// -- Granularity hardcoded for the moment.
// -- Idea is: UTb has dimensions in x (at y = 0) of about -860mm -> 860mm
// -- The indices go from 0 -> 84, and shift by -42, leading to -42 -> 42
// -- Taking the higher density of hits in the center into account, the positions of the iterators are
// -- calculated as index*index/2, where index = [ -42, 42 ], leading to
// -- -882mm -> 882mm
// -- The last element is an "end" iterator, to make sure we never go out of bound
// ==============================================================================
__host__ __device__ void fillIterators(
  UTHits& ut_hits,
  UTHitCount& ut_hit_count,
  int posLayers[4][85] )
{
    
  for(int iStation = 0; iStation < 2; ++iStation){
    for(int iLayer = 0; iLayer < 2; ++iLayer){
      int layer = 2*iStation + iLayer;
      int layer_offset = ut_hit_count.layer_offsets[layer];
      uint n_hits_layer = ut_hit_count.n_hits_layers[layer];
      
      size_t pos = 0;
      // to do: check whether there is an efficient thrust implementation for this
      fillArray( posLayers[layer], 85, pos );
      
      int bound = -42.0;
      // to do : make copysignf
      float val = std::copysign(float(bound*bound)/2.0, bound);
      
      // TODO add bounds checking
      for ( ; pos != n_hits_layer; ++pos) {
        while( ut_hits.xAtYEq0[layer_offset + pos] > val){
          posLayers[layer][bound+42] = pos;
          ++bound;
          val = std::copysign(float(bound*bound)/2.0, bound);
        }
      }
      
      fillArrayAt(
        posLayers[layer],
        42 + bound,
        85 - 42 - bound,
        n_hits_layer
      );
    }
  }
}


// ==============================================================================
// -- Finds the hits in a given layer within a certain range
// ==============================================================================
__host__ __device__ void findHits( 
  const size_t posBeg,
  const size_t posEnd,
  UTHits& ut_hits,
  uint layer_offset,
  const int i_layer,
  const float* ut_dxDy,
  const MiniState& myState, 
  const float xTolNormFact,
  const float invNormFact,
  int hitCandidatesInLayer[VeloUTTracking::max_hit_candidates_per_layer],
  int &n_hitCandidatesInLayer,
  float x_pos_layers[VeloUTTracking::n_layers][VeloUTTracking::max_hit_candidates_per_layer])
{
  const auto zInit = ut_hits.zAtYEq0[layer_offset + posBeg];
  const auto yApprox = myState.y + myState.ty * (zInit - myState.z);
  
  size_t pos = posBeg;
  while ( 
   pos <= posEnd && 
   ut_hits.isNotYCompatible( layer_offset + pos, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(xTolNormFact) )
   ) { ++pos; }

  const auto xOnTrackProto = myState.x + myState.tx*(zInit - myState.z);
  const auto yyProto =       myState.y - myState.ty*myState.z;
  
  for (int i=pos; i<posEnd; ++i) {
    const float dxDy = ut_dxDy[i_layer];
    const auto xx = ut_hits.xAt(layer_offset + i, yApprox, dxDy); 
    const auto dx = xx - xOnTrackProto;
    
    if( dx < -xTolNormFact ) continue;
    if( dx >  xTolNormFact ) break; 
    
    // -- Now refine the tolerance in Y
    if ( ut_hits.isNotYCompatible( layer_offset + i, yApprox, PrVeloUTConst::yTol + PrVeloUTConst::yTolSlope * std::abs(dx*invNormFact)) ) continue;
    
    const auto zz = ut_hits.zAtYEq0[layer_offset + i]; 
    const auto yy = yyProto +  myState.ty*zz;
    const auto xx2 = ut_hits.xAt(layer_offset + i, yy, dxDy);
        
    hitCandidatesInLayer[n_hitCandidatesInLayer] = i;
    x_pos_layers[i_layer][n_hitCandidatesInLayer] = xx2;
    
    n_hitCandidatesInLayer++;

    if ( n_hitCandidatesInLayer >= VeloUTTracking::max_hit_candidates_per_layer )
      printf("%u > %u !! \n", n_hitCandidatesInLayer, VeloUTTracking::max_hit_candidates_per_layer);
    assert( n_hitCandidatesInLayer < VeloUTTracking::max_hit_candidates_per_layer );
  }
  for ( int i_hit = 0; i_hit < n_hitCandidatesInLayer; ++i_hit ) {
    assert( hitCandidatesInLayer[i_hit] < VeloUTTracking::max_numhits_per_event );
  }
}


