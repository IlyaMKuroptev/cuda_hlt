#include "hip/hip_runtime.h"
#include "FillCandidates.cuh"
#include "VeloEventModel.cuh"
#include <cassert>

__device__ void fill_candidates_impl(
  short* h0_candidates,
  short* h2_candidates,
  const uint* module_hitStarts,
  const uint* module_hitNums,
  const float* hit_Phis,
  const uint hit_offset
) {
  // Notation is m0, m1, m2 in reverse order for each module
  // A hit in those is h0, h1, h2 respectively

  // Assign a h1 to each threadIdx.x
  const auto module_index = blockIdx.y + 2; // 48 blocks y
  const auto m1_hitNums = module_hitNums[module_index];
  for (auto i=0; i<(m1_hitNums + blockDim.x - 1) / blockDim.x; ++i) {
    const auto h1_rel_index = i*blockDim.x + threadIdx.x;

    if (h1_rel_index < m1_hitNums) {
      // Find for module module_index, hit h1_rel_index the candidates
      const auto m0_hitStarts = module_hitStarts[module_index+2] - hit_offset;
      const auto m2_hitStarts = module_hitStarts[module_index-2] - hit_offset;
      const auto m0_hitNums = module_hitNums[module_index+2];
      const auto m2_hitNums = module_hitNums[module_index-2];
      const auto h1_index = module_hitStarts[module_index] + h1_rel_index - hit_offset;

      // Calculate phi limits
      const float h1_phi = hit_Phis[h1_index];

      // Find candidates
      bool first_h0_found = false, last_h0_found = false;
      bool first_h2_found = false, last_h2_found = false;
      
      // Add h0 candidates
      for (auto h0_rel_index=0; h0_rel_index < m0_hitNums; ++h0_rel_index) {
        const unsigned short h0_index = m0_hitStarts + h0_rel_index;
        const auto h0_phi = hit_Phis[h0_index];
        const bool tolerance_condition = fabs(h1_phi - h0_phi) < VeloTracking::phi_extrapolation;

        if (!first_h0_found && tolerance_condition) {
          h0_candidates[2*h1_index] = h0_index;
          first_h0_found = true;
        }
        else if (first_h0_found && !last_h0_found && !tolerance_condition) {
          h0_candidates[2*h1_index + 1] = h0_index;
          last_h0_found = true;
          break;
        }
      }
      if (first_h0_found && !last_h0_found) {
        h0_candidates[2*h1_index + 1] = m0_hitStarts + m0_hitNums;
      }
      // In case of repeated execution, we need to populate
      // the candidates with -1 if not found
      else if (!first_h0_found) {
        h0_candidates[2*h1_index] = -1;
        h0_candidates[2*h1_index + 1] = -1;
      }

      // Add h2 candidates
      for (int h2_rel_index=0; h2_rel_index < m2_hitNums; ++h2_rel_index) {
        const unsigned short h2_index = m2_hitStarts + h2_rel_index;
        const auto h2_phi = hit_Phis[h2_index];
        const bool tolerance_condition = fabs(h1_phi - h2_phi) < VeloTracking::phi_extrapolation;

        if (!first_h2_found && tolerance_condition) {
          h2_candidates[2*h1_index] = h2_index;
          first_h2_found = true;
        }
        else if (first_h2_found && !last_h2_found && !tolerance_condition) {
          h2_candidates[2*h1_index + 1] = h2_index;
          last_h2_found = true;
          break;
        }
      }
      if (first_h2_found && !last_h2_found) {
        h2_candidates[2*h1_index + 1] = m2_hitStarts + m2_hitNums;
      }
      else if (!first_h2_found) {
        h2_candidates[2*h1_index] = -1;
        h2_candidates[2*h1_index + 1] = -1;
      }
    }
  }
}

__global__ void fill_candidates(
  uint* dev_velo_cluster_container,
  uint* dev_module_cluster_start,
  uint* dev_module_cluster_num,
  short* dev_h0_candidates,
  short* dev_h2_candidates
) {
  /* Data initialization */
  // Each event is treated with two blocks, one for each side.
  const uint event_number = blockIdx.x;
  const uint number_of_events = gridDim.x;

  // Pointers to data within the event
  const uint number_of_hits = dev_module_cluster_start[VeloTracking::n_modules * number_of_events];
  const uint* module_hitStarts = dev_module_cluster_start + event_number * VeloTracking::n_modules;
  const uint* module_hitNums = dev_module_cluster_num + event_number * VeloTracking::n_modules;
  const uint hit_offset = module_hitStarts[0];
  assert((module_hitStarts[52] - module_hitStarts[0]) < VeloTracking::max_number_of_hits_per_event);
  
  // Order has changed since SortByPhi
  const float* hit_Phis = (float*) (dev_velo_cluster_container + 4 * number_of_hits + hit_offset);
  short* h0_candidates = dev_h0_candidates + 2*hit_offset;
  short* h2_candidates = dev_h2_candidates + 2*hit_offset;

  fill_candidates_impl(
    h0_candidates,
    h2_candidates,
    module_hitStarts,
    module_hitNums,
    hit_Phis,
    hit_offset
  );
}
