#include "hip/hip_runtime.h"
#include "../include/ConsolidateTracks.cuh"


/**
 * @brief Calculates the parameters according to a root means square fit
 */
 __device__ Velo::State means_square_fit(
  Velo::Consolidated::Hits& consolidated_hits,
  const float* hit_Xs,
  const float* hit_Ys,
  const float* hit_Zs,
  const uint* hit_IDs,
  const Velo::TrackHits& track
) {
  Velo::State state;

  // Fit parameters
  float s0, sx, sz, sxz, sz2;
  float u0, uy, uz, uyz, uz2;
  s0 = sx = sz = sxz = sz2 = 0.0f;
  u0 = uy = uz = uyz = uz2 = 0.0f;
  
  // Iterate over hits
  for (unsigned short h=0; h<track.hitsNum; ++h) {
    const auto hit_index = track.hits[h];
    const auto x = hit_Xs[hit_index];
    const auto y = hit_Ys[hit_index];
    const auto z = hit_Zs[hit_index];
    
    const auto wx = VeloTracking::param_w;
    const auto wx_t_x = wx * x;
    const auto wx_t_z = wx * z;
    s0 += wx;
    sx += wx_t_x;
    sz += wx_t_z;
    sxz += wx_t_x * z;
    sz2 += wx_t_z * z;

    const auto wy = VeloTracking::param_w;
    const auto wy_t_y = wy * y;
    const auto wy_t_z = wy * z;
    u0 += wy;
    uy += wy_t_y;
    uz += wy_t_z;
    uyz += wy_t_y * z;
    uz2 += wy_t_z * z;
  }

  {
    // Calculate tx, ty and backward
    const auto dens = 1.0f / (sz2 * s0 - sz * sz);
    state.tx = (sxz * s0 - sx * sz) * dens;
    state.x = (sx * sz2 - sxz * sz) * dens;

    const auto denu = 1.0f / (uz2 * u0 - uz * uz);
    state.ty = (uyz * u0 - uy * uz) * denu;
    state.y = (uy * uz2 - uyz * uz) * denu;

    state.z = -(state.x * state.tx + state.y * state.ty) / (state.tx * state.tx + state.ty * state.ty);

    state.backward = state.z > consolidated_hits.z[0];
  }

  {
    // Covariance
    const auto m00 = s0;
    const auto m11 = u0;
    const auto m20 = sz - state.z * s0;
    const auto m31 = uz - state.z * u0;
    const auto m22 = sz2 - 2 * state.z * sz + state.z * state.z * s0;
    const auto m33 = uz2 - 2 * state.z * uz + state.z * state.z * u0;
    const auto den20 = 1.0f / (m22 * m00 - m20 * m20);
    const auto den31 = 1.0f / (m33 * m11 - m31 * m31);

    state.c00 = m22 * den20;
    state.c20 = -m20 * den20;
    state.c22 = m00 * den20;
    state.c11 = m33 * den31;
    state.c31 = -m31 * den31;
    state.c33 = m11 * den31;
  }

  {
    //=========================================================================
    // Chi2 / degrees-of-freedom of straight-line fit
    //=========================================================================
    float ch = 0.0f;
    int nDoF = -4;
    for (uint h=0; h<track.hitsNum; ++h) {
      const auto z = consolidated_hits.z[h];

      const auto x = state.x + state.tx * z;
      const auto y = state.y + state.ty * z;

      const auto dx = x - consolidated_hits.x[h];
      const auto dy = y - consolidated_hits.y[h];
      
      ch += dx * dx * VeloTracking::param_w + dy * dy * VeloTracking::param_w;

      // Nice :)
      // TODO: We can get rid of the X and Y read here
      // float sum_w_xzi_2 = CL_VeloTracking::param_w * x; // for each hit
      // float sum_w_xi_2 = CL_VeloTracking::param_w * hit_Xs[hitno]; // for each hit
      // ch = (sum_w_xzi_2 - sum_w_xi_2) + (sum_w_yzi_2 - sum_w_yi_2);

      nDoF += 2;
    }
    state.chi2 = ch / nDoF; 
  }

  state.x = state.x + state.tx * state.z;
  state.y = state.y + state.ty * state.z;

  return state;
}

__global__ void consolidate_tracks(
  int* dev_atomics_storage,
  const Velo::TrackHits* dev_tracks,
  uint* dev_velo_track_hit_number,
  uint* dev_velo_cluster_container,
  uint* dev_module_cluster_start,
  uint* dev_module_cluster_num,
  uint* dev_velo_track_hits,
  uint* dev_velo_states
) {
  const uint number_of_events = gridDim.x;
  const uint event_number = blockIdx.x;
  const Velo::TrackHits* event_tracks = dev_tracks + event_number * VeloTracking::max_tracks;

  // Consolidated datatypes
  const Velo::Consolidated::Tracks velo_tracks {(uint*) dev_atomics_storage, dev_velo_track_hit_number, event_number, number_of_events};
  Velo::Consolidated::States velo_states {dev_velo_states, velo_tracks.total_number_of_tracks};

  const uint number_of_tracks_event = velo_tracks.number_of_tracks(event_number);
  const uint event_tracks_offset = velo_tracks.tracks_offset(event_number);

  // Pointers to data within event
  const uint number_of_hits = dev_module_cluster_start[VeloTracking::n_modules * number_of_events];
  const uint* module_hitStarts = dev_module_cluster_start + event_number * VeloTracking::n_modules;
  const uint hit_offset = module_hitStarts[0];
  
  // Order has changed since SortByPhi
  const float* hit_Ys   = (float*) (dev_velo_cluster_container + hit_offset);
  const float* hit_Zs   = (float*) (dev_velo_cluster_container + number_of_hits + hit_offset);
  const float* hit_Xs   = (float*) (dev_velo_cluster_container + 5 * number_of_hits + hit_offset);
  const uint32_t* hit_IDs = (uint32_t*) (dev_velo_cluster_container + 2 * number_of_hits + hit_offset);

  for (uint i=threadIdx.x; i<number_of_tracks_event; i+=blockDim.x) {
    Velo::Consolidated::Hits consolidated_hits = velo_tracks.get_hits(dev_velo_track_hits, i);
    const Velo::TrackHits track = event_tracks[i];

    auto populate = [&track] (uint32_t* __restrict__ a, uint32_t* __restrict__ b) {
      for (int i=0; i<track.hitsNum; ++i) {
        const auto hit_index = track.hits[i];
        a[i] = b[hit_index];
      }
    };

    populate((uint32_t*) consolidated_hits.x, (uint32_t*) hit_Xs);
    populate((uint32_t*) consolidated_hits.y, (uint32_t*) hit_Ys);
    populate((uint32_t*) consolidated_hits.z, (uint32_t*) hit_Zs);
    populate((uint32_t*) consolidated_hits.LHCbID, (uint32_t*) hit_IDs);

    // Calculate and store fit in consolidated container
    Velo::State beam_state = means_square_fit(
      consolidated_hits,
      hit_Xs,
      hit_Ys,
      hit_Zs,
      hit_IDs,
      track
    );

    velo_states.set(event_tracks_offset + i, beam_state);
  }
}
